#include "hip/hip_runtime.h"
/*
 	Main application
	Author : Bulat, 2015

*/
//#include "graph.cuh"
#include "apsp.cuh"
#include "graph.cuh"
#include "headers.h"


/*
*	Expand array according to it's from and to values
*	Input : device_ptr<vertex> expanded_array
*			device_ptr<vertex> position_current_level
*			device_ptr<vertex> current_ending_offset
*	Out: 	sorted and normalized expanded_array
*/
__global__  void expander(
	device_ptr<vertex> current_vertex, device_ptr<vertex> temp_from, device_ptr<vertex> temp_to,
	device_ptr<vertex> full_vertex_array, device_ptr<vertex> full_edge_array,
	device_ptr<vertex> position_in_array,
	device_ptr<vertex> expanded_array, device_ptr<vertex> from_vertex_array,
	int number_edges_to_process, int number_of_vertex,
	int current_level,
	device_ptr<vertex> vertex_offset
	)
{

	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int offset_to_put_exp_array = 0;
	/*
	*	For searching in full_edge_list if already discovered
	*/
	if (idx != 0)
	{
		offset_to_put_exp_array = position_in_array[idx - 1]; // reserved position in expanded array
	}
	/*
	*	Copy to expanded array if the edge is unique (was not discovered previously, not equal to vertex itself)
	*	Result:			1 2 1 .... (expanded edges)
	*/
	thrust::device_ptr<vertex> current_position =
 	thrust::copy_if(thrust::device,
		thrust::make_permutation_iterator(full_edge_array, thrust::make_counting_iterator<vertex>(temp_from[idx])),
		thrust::make_permutation_iterator(full_edge_array, thrust::make_counting_iterator<vertex>(temp_to[idx])),
		expanded_array + offset_to_put_exp_array,
		unique_edge(full_vertex_array, full_edge_array, current_vertex[idx],
		number_of_vertex, current_level));

	int planed_size = temp_to[idx] - temp_from[idx];
	int real_size = thrust::distance(expanded_array + offset_to_put_exp_array, current_position);
	int starting = current_vertex[idx];
	// TODO : check real size value

	/*
	*	Expand the current processing vertex to the size *real size*;
	*			Result : 0 0 0 1 1 ... (the vertex from expanded)
	*/
	thrust::copy(thrust::device,
		thrust::make_constant_iterator(starting),
		thrust::make_constant_iterator(starting) + real_size,
		from_vertex_array + offset_to_put_exp_array);
	vertex* k = thrust::raw_pointer_cast(vertex_offset + starting);
	atomicAdd(k, real_size);

	if (planed_size != real_size)
		if (idx != 0)
	{
			thrust::transform(thrust::device, position_in_array + idx - 1,
			position_in_array + number_edges_to_process,
			position_in_array + idx -1, minus_value(planed_size - real_size));
	}
		else
		{
			thrust::transform(thrust::device, position_in_array,
				position_in_array + number_edges_to_process, position_in_array, minus_value(planed_size - real_size));
		}
}


/*
*	Sorting function for each vertex offset position. Initial sort (not for L > 1)
*	Input: device_ptr full_edge_array
*		   device_ptr full_vertex_array
*	Out : Sorted full_edge_array
*/
__global__ void sorter(thrust::device_ptr<vertex> full_edge_array,
						thrust::device_ptr<vertex> full_vertex_array)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int starting_point = 0;
	if (idx != 0)
	{
		starting_point = full_vertex_array[idx - 1];
	}
	int ending_point = full_vertex_array[idx];
	// sort
	thrust::sort(thrust::device, full_edge_array + starting_point, full_edge_array + ending_point);

}

/*
*	Sort and remove duplicates in edge_array
*	Input : device_ptr<vertex> expanded_array
*			device_ptr<vertex> position_current_level
*			device_ptr<vertex> current_ending_offset
*	Out: 	sorted and normalized expanded_array
*/
__global__ void unifier(
	device_ptr<vertex> expanded_array,
	device_ptr<vertex> positions_vertex_current_level,
	device_ptr<vertex> current_ending_offset)
	{
		int idx = blockIdx.x*blockDim.x + threadIdx.x;
		int start_point = 0;

		if (idx != 0)
		{
			start_point = positions_vertex_current_level[idx - 1];

		}
		int end_point = positions_vertex_current_level[idx];
		if (end_point > start_point)
		{
			thrust::sort(thrust::device, expanded_array + start_point, expanded_array + end_point);
			// remove dublicates
			thrust::device_ptr<vertex> current_position =
				thrust::unique(thrust::device, expanded_array + start_point, expanded_array + end_point);
			vertex real_size = thrust::distance(expanded_array + start_point, current_position);
			current_ending_offset[idx] = real_size;
		}
		else
		{
			current_ending_offset[idx] = 0;
		}


	}


__global__ void edge_copier(
	device_ptr<vertex> expanded_array,
	device_ptr<vertex> positions_vertex_current_level,
	device_ptr<vertex> current_ending_offset,
	device_ptr<vertex> full_vertex_array,
	device_ptr<vertex> full_edge_array,
	int L_VALUE,
	int number_of_vertex
	)

{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int start_point = 0;

	if (idx != 0)
	{
		start_point = positions_vertex_current_level[idx - 1];

	}
	int end_point = start_point + current_ending_offset[idx];

	int edge_put_list_start = full_vertex_array[L_VALUE *number_of_vertex + idx - 1];

	thrust::copy(thrust::device, expanded_array + start_point, expanded_array + end_point, full_edge_array + edge_put_list_start);
}

__global__ void opacity_former(
	device_ptr<vertex> from,
	device_ptr<vertex> to,
	device_ptr<vertex> degree_count,
	device_ptr<opacity> opacity_matrix,
	int max_degree
	)
	{
		// TODO: Not sure about this
		int i = blockIdx.x*blockDim.x + threadIdx.x;
		opacity min = degree_count[from[i] - 1] * degree_count[to[i] - 1];
		if (degree_count[from[i] - 1] == degree_count[to[i] - 1])
			min = degree_count[from[i] - 1];
		opacity* k = thrust::raw_pointer_cast(opacity_matrix + max_degree*(from[i] - 1) + (to[i] - 1));
		opacity added_value = 1.0/ (2.0 * min);
		atomicAdd(k, added_value);

	}

/*
*	By finding shortest paths, form to L_VALUE level
*/
void form_full_level_graph(Graph graph)
{

	for (int current_level = 1; current_level < graph.L_VALUE; current_level++)
	{
	int starting_point = 0;
	int ending_point = graph.full_vertex_array[current_level * graph.number_of_vertex - 1];
	if (current_level != 1)
	{
		starting_point = graph.full_vertex_array[(current_level - 1) * graph.number_of_vertex - 1]; // previous last element
	}

	vertex number_edges_to_process = ending_point- starting_point;
	// Value to add as an offset (previous end - current begin)
	int added_offset = starting_point;

	device_ptr<vertex> temp_to =  device_malloc<vertex>(number_edges_to_process);
	device_ptr<vertex> temp_from =  device_malloc<vertex>(number_edges_to_process);

	/* Form temp to an temp from vector from edge arrray */
	thrust::copy(thrust::device, graph.full_edge_array + starting_point,
	graph.full_edge_array + ending_point,
	temp_to);

	thrust::copy(thrust::device,
		graph.full_edge_array + starting_point, graph.full_edge_array + ending_point,
		 temp_from);
		//
		thrust::transform(thrust::device,
			temp_from, temp_from + number_edges_to_process,
			temp_from, previous_el(current_level * graph.number_of_vertex + 1));

	/* Store begining and ending */
	thrust::copy(
		thrust::device,
		thrust::make_permutation_iterator(graph.full_vertex_array, temp_to),
		thrust::make_permutation_iterator(graph.full_vertex_array, temp_to + number_edges_to_process),
		temp_to);

	thrust::copy(
		thrust::device,
		thrust::make_permutation_iterator(graph.full_vertex_array, temp_from),
		thrust::make_permutation_iterator(graph.full_vertex_array,
		temp_from + number_edges_to_process), temp_from);


	/*
	*	Array of vertex, from which we will expand. Proces vertex
	*/
	device_ptr<vertex>  process_vetxes = device_malloc<vertex>(number_edges_to_process+1);
	/* Find all breaking points. 0 0 1 0 0 0 1 ... */
	thrust::transform(
		thrust::device,
		thrust::make_counting_iterator<vertex>(added_offset),
		thrust::make_counting_iterator<vertex>(number_edges_to_process + added_offset),
		process_vetxes,
		replacer(graph.full_vertex_array + (current_level-1)* graph.number_of_vertex, graph.number_of_vertex));
	/*
		Sum all previous results (sum of breaking points).
		Result: 0 0 1 1 1 1 2 2 2 (vertex array)
	*/
		thrust::inclusive_scan(thrust::device, process_vetxes,
									process_vetxes + number_edges_to_process, process_vetxes);

	/*
		Offset array, step 1:
		Result <- Temp_TO - Temp_FROM
	*/
		device_ptr<vertex>  position_in_array = device_malloc<vertex>(number_edges_to_process);

		thrust::transform(
			thrust::device,
			make_zip_iterator(thrust::make_tuple(temp_from, temp_to)),
			make_zip_iterator(thrust::make_tuple(temp_from + number_edges_to_process,
												temp_to + number_edges_to_process)),
			position_in_array,
			counter());
	/*
		Forming offset array from process number, step 2:
		2 4 4 => 2 6 10
	*/
	thrust::inclusive_scan(thrust::device, position_in_array,
					 		    position_in_array + number_edges_to_process,
							    position_in_array);




	device_ptr<vertex> expanded_array = device_malloc<vertex>(position_in_array[number_edges_to_process - 1]);
	thrust::fill(thrust::device, expanded_array, expanded_array + position_in_array[number_edges_to_process - 1], -1);
	// process number contains the maximum needed memory to store if all vertexes are unique
	device_ptr<vertex> from_vertex_array = device_malloc<vertex>(position_in_array[number_edges_to_process - 1]);
	thrust::fill(thrust::device, from_vertex_array, from_vertex_array + position_in_array[number_edges_to_process - 1], -1);

	int prev_max_position = position_in_array[number_edges_to_process - 1];
	//  Expand array on one level
	//	Can contain non unique values

	int grid_size = number_edges_to_process;
	thrust::device_ptr<vertex> positions_vertex_current_level = thrust::device_malloc<vertex>(graph.number_of_vertex);
	thrust::fill(thrust::device, positions_vertex_current_level, positions_vertex_current_level + graph.number_of_vertex, 0);

	expander<<< 1, grid_size >>>(
		process_vetxes, temp_from, temp_to,
		graph.full_vertex_array, graph.full_edge_array,
		position_in_array,
		expanded_array, from_vertex_array,
		number_edges_to_process,
		graph.number_of_vertex,
		current_level,
		positions_vertex_current_level);
	//hipDeviceSynchronize();
	hipDeviceSynchronize();

	device_free(temp_from);
	device_free(temp_to);
	device_free(process_vetxes);

	/*
	*	Remove empty, non used data
		*/
	thrust::remove(thrust::device, expanded_array, expanded_array + prev_max_position, -1);
	thrust::remove(thrust::device, from_vertex_array, from_vertex_array + prev_max_position, -1);

	/*
	*	Form vertex offset list
	*/
	//	int gridsize = position_in_array[number_edges_to_process - 1];

	// STEP 2: Forming offset
	thrust::inclusive_scan(thrust::device, positions_vertex_current_level,
							positions_vertex_current_level + graph.number_of_vertex, positions_vertex_current_level);

	thrust::device_ptr<vertex> vertex_ending_offsets = thrust::device_malloc<vertex>(graph.number_of_vertex);
	unifier <<<1, graph.number_of_vertex >>>( expanded_array, positions_vertex_current_level, vertex_ending_offsets);


	thrust::device_ptr<vertex> position_in_edge_list = thrust::device_malloc<vertex>(graph.number_of_vertex);

	cout << endl;
	hipDeviceSynchronize();

	thrust::copy(thrust::device, vertex_ending_offsets, vertex_ending_offsets + graph.number_of_vertex,
		graph.full_vertex_array + current_level * graph.number_of_vertex);



	thrust::inclusive_scan(thrust::device, graph.full_vertex_array + current_level * graph.number_of_vertex - 1,
		graph.full_vertex_array + (current_level + 1) * graph.number_of_vertex, graph.full_vertex_array + current_level * graph.number_of_vertex - 1);


	grid_size = graph.number_of_vertex;

	edge_copier<<<1, grid_size>>>(
		expanded_array,
		positions_vertex_current_level,
		vertex_ending_offsets,
		graph.full_vertex_array,
		graph.full_edge_array,
		current_level,
		graph.number_of_vertex);
	hipDeviceSynchronize();
	device_free(expanded_array);
	device_free(positions_vertex_current_level);
	device_free(vertex_ending_offsets);


	}
}



void ordering_function(Graph graph)
{
	sorter<<<1, graph.number_of_vertex>>>(graph.full_edge_array, graph.full_vertex_array);
}


/*********************************
*	L opacity matrix calculation
*********************************/

void calc_L_opacity(Graph graph)
{
	for (int i = 1; i <= graph.L_VALUE; i++)
	{
		// full_edge_array - here we store all adjasent

		// Forming indexes (from vertex)
		int starting_point = 0;
		int ending_point = graph.full_vertex_array[(i)*graph.number_of_vertex - 1];

		if (i != 1)

		{
			starting_point = graph.full_vertex_array[(i - 1)* graph.number_of_vertex - 1];
		}

		vertex N = ending_point - starting_point;
		device_ptr<vertex> from = device_malloc<vertex>(N);

		/*
		*	Expanding indexes. Finding break points
		*	Example: 0 1 2 3 4 .. 20 => 0 0 1 0 0 0 1 ...
		*/

		thrust::transform(
			thrust::device,
			thrust::make_counting_iterator<vertex>(starting_point),
			thrust::make_counting_iterator<vertex>(ending_point),
			from, replacer(graph.full_vertex_array + (i-1)* graph.number_of_vertex, graph.number_of_vertex)
			);



		//	from[0] = full_vertex_array[(number_of_vertex-1)*(i-1)];
		/*
		*	Transorming into indexes:
		*	Example:	0 0 1 0 0 0 1 => 0 0 1 1 1 1 2 2 2 ..
		*/

		thrust::inclusive_scan(thrust::device, from, from + N , from);


		/*
		*	Transforming from indexes into degrees:
		*	Example:  0 0 1 1 1 1 2 2 2.. => 2 2 4 4 4 4 ...
		*/

		thrust::transform(
			thrust::device,
			thrust::make_permutation_iterator(graph.vertex_degrees, from),
			thrust::make_permutation_iterator(graph.vertex_degrees, from + N),
			from, thrust::identity<vertex>());

		/*
		*	To vector. Transform edge list into degree list =>  similar techno
		*
		*/


		thrust::device_ptr<vertex> to = device_malloc<vertex>(N);
		//	auto iter_begin = thrust::make_transform_iterator(full_edge_array.begin(), minus_one());
		//	auto iter_end =   thrust::make_transform_iterator(full_edge_array.begin() + N, minus_one());

		thrust::copy(thrust::device, graph.full_edge_array + starting_point, graph.full_edge_array + ending_point, to);

		thrust::transform(
			thrust::device,
			thrust::make_permutation_iterator(graph.vertex_degrees, to),
			thrust::make_permutation_iterator(graph.vertex_degrees, to + N),
			to, thrust::identity<vertex>());

		/*
		*  Find max and min in zip iterator of to - from pairs
		*/


		thrust::transform(
			thrust::device,
			thrust::make_zip_iterator(thrust::make_tuple(from, to)),
			thrust::make_zip_iterator(thrust::make_tuple(from + N, to + N)),
			thrust::make_zip_iterator(thrust::make_tuple(from, to)),
			min_max_transform());

		/*
		* 	Opacity  matrix forming. Now it is n^ 2 memory TODO: IN PARALLEL using cuda kernel
		* 	Assumptions !!: Not optimum for undericted (div 2).
		* 	Problem with same degree. Example: {4 = > 4} - must count only degree of one.
		*/
		int gridsize = N;
		opacity_former<<<1, gridsize>>>(from, to, graph.degree_count, graph.opacity_matrix, graph.max_degree);

		/*
		* Sort by key. Indexes (values) and degrees (keys)
		*/

		/*
		* 	Reduce by key. Count how many pairs we have. 0 0 3 3
		*/

	}
}


int main(int argc, char* argv[])
{

	Graph graph;
	cout << "Converting to " << endl;
	int l_value = std::atoi(argv[1]);
	cout << l_value << " L value" << endl;
	graph.L_VALUE = l_value;
	graph.init_test_graph(); // Reading graph from the file in COO format
	graph.print_coo_graph();
	graph.convert_to_CSR();
	graph.print_csr_graph();
	ordering_function(graph);



	UINT wTimerRes = 0;
	bool init = InitMMTimer(wTimerRes);
	DWORD startTime = timeGetTime();

		form_full_level_graph(graph);

	// BFS
	//graph.single_bfs(2);
//	graph.form_full_level_graph();
//	graph.print_csr_graph();
		calc_L_opacity(graph);
  	graph.print_opacity_matrix();

	unsigned int endTime = timeGetTime();
	unsigned int gpu_time = unsigned int(endTime - startTime);
	printf("GPU Timing(including all device-host, host-device copies, device allocations and freeing of device memory): %dms\n\n", gpu_time);
	DestroyMMTimer(wTimerRes, init);

	graph.print_csr_graph();


	return 0;
}
