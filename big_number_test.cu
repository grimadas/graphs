#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_vector_types.h>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/equal.h>

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

#define DSIZE (32*1048576)

struct sort_f4_w
{
  __host__ __device__
  bool operator()(const float4 &a, const float4 &b) const {
    return (a.w < b.w);}
};
// functor to extract the .w element from a float4
struct f4_to_fw : public thrust::unary_function<float4, float>
{
  __host__ __device__
  float operator()(const float4 &a) const {
    return a.w;}
};
// functor to extract the .x element from a float4
struct f4_to_fx : public thrust::unary_function<float4, float>
{
  __host__ __device__
  float operator()(const float4 &a) const {
    return a.x;}
};


bool validate(thrust::device_vector<float4> &d1, thrust::device_vector<float4> &d2){
  return thrust::equal(thrust::make_transform_iterator(d1.begin(), f4_to_fx()), thrust::make_transform_iterator(d1.end(), f4_to_fx()), thrust::make_transform_iterator(d2.begin(), f4_to_fx()));
}


int main(){
  unsigned long long t1_time, t2_time;
  float4 *mydata = new float4[DSIZE];
  for (int i = 0; i < DSIZE; i++){
    mydata[i].x = i;
    mydata[i].y = i;
    mydata[i].z = i;
    mydata[i].w = rand()/(float)RAND_MAX;}

  thrust::host_vector<float4>   h_data(mydata, mydata+DSIZE);
  // do once as a warm-up run, then report timings on second run
  for (int i = 0; i < 2; i++){
    thrust::device_vector<float4> d_data1 = h_data;
    thrust::device_vector<float4> d_data2 = h_data;

  // first time sort using typical thrust approach
    t1_time = dtime_usec(0);
    thrust::sort(d_data1.begin(), d_data1.end(), sort_f4_w());
    hipDeviceSynchronize();
    t1_time = dtime_usec(t1_time);
  // now extract keys and create index values, sort, then rearrange
    t2_time = dtime_usec(0);
    thrust::device_vector<float> keys(DSIZE);
    thrust::device_vector<int> vals(DSIZE);
    thrust::copy(thrust::make_transform_iterator(d_data2.begin(), f4_to_fw()), thrust::make_transform_iterator(d_data2.end(), f4_to_fw()), keys.begin());
    thrust::sequence(vals.begin(), vals.end());
    thrust::sort_by_key(keys.begin(), keys.end(), vals.begin());
    thrust::device_vector<float4> result(DSIZE);
    thrust::copy(thrust::make_permutation_iterator(d_data2.begin(), vals.begin()), thrust::make_permutation_iterator(d_data2.begin(), vals.end()), result.begin());
    hipDeviceSynchronize();
    t2_time = dtime_usec(t2_time);
    if (!validate(d_data1, result)){
      std::cout << "Validation failure " << std::endl;
      }
    }
  std::cout << "thrust t1 time: " << t1_time/(float)USECPSEC << "s, t2 time: " << t2_time/(float)USECPSEC << std::endl;
}
